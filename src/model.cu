#include "hip/hip_runtime.h"
// fastEIT
//
// Copyright (C) 2012  Patrik Gebhardt
// Contact: patrik.gebhardt@rub.de

// redefine extern c
#define LINALGCU_EXTERN_C extern "C"

#include <stdlib.h>
#include <stdio.h>
#include "../include/fasteit.h"

// reduce connectivity and elementalResidual matrix
__global__ void reduce_matrix_kernel(linalgcuMatrixData_t* matrix,
    linalgcuMatrixData_t* intermediateMatrix, linalgcuColumnId_t* systemMatrixColumnIds,
    linalgcuSize_t rows, linalgcuSize_t density) {
    // get ids
    linalgcuSize_t row = blockIdx.x * blockDim.x + threadIdx.x;
    linalgcuSize_t column = blockIdx.y * blockDim.y + threadIdx.y;

    // get column id
    linalgcuColumnId_t columnId = systemMatrixColumnIds[row * LINALGCU_SPARSE_SIZE + column];

    // check column id
    if (columnId == -1) {
        return;
    }

    // reduce matrices
    for (int k = 0; k < density; k++) {
        matrix[row + (column + k * LINALGCU_SPARSE_SIZE) * rows] =
            intermediateMatrix[row + (columnId + k * rows) * rows];
    }
}

// reduce matrix
LINALGCU_EXTERN_C
linalgcuError_t fasteit_model_reduce_matrix(fasteitModel_t self, linalgcuMatrix_t matrix,
    linalgcuMatrix_t intermediateMatrix, linalgcuSize_t density, hipStream_t stream) {
    // check input
    if ((self == NULL) || (matrix == NULL) || (intermediateMatrix == NULL)) {
        return LINALGCU_ERROR;
    }

    // block size
    dim3 blocks(matrix->rows / LINALGCU_BLOCK_SIZE, 1);
    dim3 threads(LINALGCU_BLOCK_SIZE, LINALGCU_BLOCK_SIZE);

    // reduce matrix
    reduce_matrix_kernel<<<blocks, threads, 0, stream>>>(
        matrix->deviceData, intermediateMatrix->deviceData,
        self->systemMatrix2D->columnIds, matrix->rows,
        density);

    return LINALGCU_SUCCESS;
}

// update matrix kernel
__global__ void update_matrix_kernel(linalgcuMatrixData_t* matrixValues,
    linalgcuColumnId_t* matrixColumnIds, linalgcuColumnId_t* columnIds,
    linalgcuMatrixData_t* connectivityMatrix, linalgcuMatrixData_t* elementalMatrix,
    linalgcuMatrixData_t* gamma, linalgcuMatrixData_t sigmaRef,
    linalgcuSize_t rows, linalgcuSize_t density) {
    // get ids
    linalgcuSize_t row = blockIdx.x * blockDim.x + threadIdx.x;
    linalgcuSize_t column = blockIdx.y * blockDim.y + threadIdx.y;

    // calc residual matrix element
    linalgcuMatrixData_t value = 0.0f;
    linalgcuColumnId_t elementId = -1;
    for (int k = 0; k < density; k++) {
        // get element id
        elementId = (linalgcuColumnId_t)connectivityMatrix[row +
            (column + k * LINALGCU_SPARSE_SIZE) * rows];

        value += elementId != -1 ? elementalMatrix[row +
            (column + k * LINALGCU_SPARSE_SIZE) * rows] *
            sigmaRef * exp10f(gamma[elementId] / 10.0f) : 0.0f;
    }

    // set residual matrix element
    matrixValues[row * LINALGCU_SPARSE_SIZE + column] = value;
}

// update matrix
LINALGCU_EXTERN_C
linalgcuError_t fasteit_model_update_matrix(fasteitModel_t self,
    linalgcuSparseMatrix_t matrix, linalgcuMatrix_t elementalMatrix, linalgcuMatrix_t gamma,
    hipStream_t stream) {
    // check input
    if ((self == NULL) || (matrix == NULL) || (elementalMatrix == NULL) || (gamma == NULL)) {
        return LINALGCU_ERROR;
    }

    // dimension
    dim3 threads(LINALGCU_BLOCK_SIZE, LINALGCU_BLOCK_SIZE);
    dim3 blocks(matrix->rows / LINALGCU_BLOCK_SIZE, 1);

    // execute kernel
    update_matrix_kernel<<<blocks, threads, 0, stream>>>(
        matrix->values, matrix->columnIds, self->systemMatrix2D->columnIds,
        self->connectivityMatrix->deviceData, elementalMatrix->deviceData,
        gamma->deviceData, self->sigmaRef, self->connectivityMatrix->rows,
        matrix->density);

    return LINALGCU_SUCCESS;
}
