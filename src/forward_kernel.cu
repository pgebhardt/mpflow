#include "hip/hip_runtime.h"
// fastEIT
//
// Copyright (C) 2012  Patrik Gebhardt
// Contact: patrik.gebhardt@rub.de

#include <cstdlib>
#include <cstdio>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "fasteit/cuda_error.h"

#include "fasteit/dtype.h"
#include "fasteit/constants.h"
#include "fasteit/forward_kernel.h"

// calc jacobian kernel
template <
    int nodes_per_element
>
static __global__ void calcJacobianKernel(const fastEIT::dtype::real* drivePhi,
    const fastEIT::dtype::real* measurmentPhi, const fastEIT::dtype::index* connectivityMatrix,
    const fastEIT::dtype::real* elementalJacobianMatrix, const fastEIT::dtype::real* gamma,
    fastEIT::dtype::real sigmaRef, fastEIT::dtype::size rows, fastEIT::dtype::size columns,
    fastEIT::dtype::size phiRows, fastEIT::dtype::size elementCount,
    fastEIT::dtype::size driveCount, fastEIT::dtype::size measurmentCount, bool additiv,
    fastEIT::dtype::real* jacobian) {
    // get id
    fastEIT::dtype::index row = blockIdx.x * blockDim.x + threadIdx.x;
    fastEIT::dtype::index column = blockIdx.y * blockDim.y + threadIdx.y;

    // check column
    if (column >= elementCount) {
        return;
    }

    // calc measurment and drive id
    fastEIT::dtype::size roundMeasurmentCount = (
        (measurmentCount + fastEIT::matrix::block_size - 1) /
        fastEIT::matrix::block_size) *
        fastEIT::matrix::block_size;
    fastEIT::dtype::size measurmentId = row % roundMeasurmentCount;
    fastEIT::dtype::size driveId = row / roundMeasurmentCount;

    // variables
    fastEIT::dtype::real dPhi[nodes_per_element], mPhi[nodes_per_element];
    fastEIT::dtype::index index;

    // get data
    for (fastEIT::dtype::index i = 0; i < nodes_per_element; i++) {
        index = connectivityMatrix[column + i * columns];
        dPhi[i] = driveId < driveCount ? drivePhi[index + driveId * phiRows] : 0.0f;
        mPhi[i] = measurmentId < measurmentCount ? measurmentPhi[index +
            measurmentId * phiRows] : 0.0f;
    }

    // calc matrix element
    fastEIT::dtype::real element = 0.0f;
    for (fastEIT::dtype::index i = 0; i < nodes_per_element; i++) {
        for (fastEIT::dtype::index j = 0; j < nodes_per_element; j++) {
            element += dPhi[i] * mPhi[j] * elementalJacobianMatrix[column +
                (i + j * nodes_per_element) * columns];
        }
    }

    // diff sigma to gamma
    element *= sigmaRef * exp10f(gamma[column] / 10.0f) / 10.0f;

    // set matrix element
    if (additiv == true) {
        jacobian[row + column * rows] += element;
    }
    else {
        jacobian[row + column * rows] = element;
    }
}

// calc jacobian kernel wrapper
template <
    int nodes_per_element
>
void fastEIT::forwardKernel::calcJacobian(dim3 blocks, dim3 threads, hipStream_t stream,
    const dtype::real* drive_phi, const dtype::real* measurment_phi,
    const dtype::index* connectivity_matrix, const dtype::real* elemental_jacobian_matrix,
    const dtype::real* gamma, dtype::real sigma_ref, dtype::size rows, dtype::size columns,
    dtype::size phi_rows, dtype::size element_count, dtype::size drive_count,
    dtype::size measurment_count, bool additiv, dtype::real* jacobian) {
    // call cuda kernel
    calcJacobianKernel<nodes_per_element><<<blocks, threads, 0, stream>>>(
        drive_phi, measurment_phi, connectivity_matrix, elemental_jacobian_matrix, gamma,
        sigma_ref, rows, columns, phi_rows, element_count, drive_count,
        measurment_count, additiv, jacobian);

    CudaCheckError();
}

// template specialisation
template void fastEIT::forwardKernel::calcJacobian<3>(dim3, dim3, hipStream_t,
    const dtype::real*, const dtype::real*, const dtype::index*, const dtype::real*,
    const dtype::real*, dtype::real, dtype::size, dtype::size, dtype::size, dtype::size,
    dtype::size, dtype::size, bool, dtype::real*);
