#include "hip/hip_runtime.h"
// fastEIT
//
// Copyright (C) 2012  Patrik Gebhardt
// Contact: patrik.gebhardt@rub.de

#include "../include/fasteit.hpp"

// namespaces
using namespace fastEIT;
using namespace std;

// add scalar kernel
__global__ void addScalarKernel(dtype::real* vector, dtype::real* scalar,
    dtype::size vectorRows, dtype::size rows, dtype::size columns) {
    // get ids
    dtype::index row = blockIdx.x * blockDim.x + threadIdx.x;
    dtype::index column = blockIdx.y * blockDim.y + threadIdx.y;

    // add data
    vector[row + column * vectorRows] += row < rows && column < columns ? scalar[column * vectorRows] : 0.0f;
}

// add scalar
void Conjugate::addScalar(Matrix<dtype::real>& vector,
    Matrix<dtype::real>& scalar, dtype::size rows, dtype::size columns, hipStream_t stream) {
    // kernel dimension
    dim3 global(vector.rows() / Matrix<dtype::real>::blockSize, vector.columns() == 1 ? 1 :
        vector.columns() / Matrix<dtype::real>::blockSize);
    dim3 local(Matrix<dtype::real>::blockSize, vector.columns() == 1 ? 1 : Matrix<dtype::real>::blockSize);

    // execute kernel
    addScalarKernel<<<global, local, 0, stream>>>(vector.deviceData(), scalar.deviceData(),
        vector.rows(), rows, columns);
}

// update vector
__global__ void updateVectorKernel(dtype::real* result, dtype::real* x1, dtype::real sign,
    dtype::real* x2, dtype::real* r1, dtype::real* r2, dtype::size rows) {
    // get ids
    dtype::index row = blockIdx.x * blockDim.x + threadIdx.x;
    dtype::index column = blockIdx.y * blockDim.y + threadIdx.y;

    // calc value
    result[row + column * rows] = r2[column * rows] != 0.0f ? x1[row + column * rows] + sign * x2[row + column * rows] *
        r1[column * rows] / r2[column * rows] : 0.0f;
}

// update vector
void Conjugate::updateVector(Matrix<dtype::real>& result,
    Matrix<dtype::real>& x1, dtype::real sign, Matrix<dtype::real>& x2,
    Matrix<dtype::real>& r1, Matrix<dtype::real>& r2, hipStream_t stream) {
    // kernel dimension
    dim3 global(result.rows() / Matrix<dtype::real>::blockSize, result.columns() == 1 ? 1 :
        result.columns() / Matrix<dtype::real>::blockSize);
    dim3 local(Matrix<dtype::real>::blockSize, result.columns() == 1 ? 1 : Matrix<dtype::real>::blockSize);

    // execute kernel
    updateVectorKernel<<<global, local, 0, stream>>>(result.deviceData(),
        x1.deviceData(), sign, x2.deviceData(), r1.deviceData(), r2.deviceData(), result.rows());
}

// gemv kernel
__global__ void gemvKernel(dtype::real* matrix, dtype::real* vector,
    dtype::real* result, dtype::size rows) {
    // get ids
    dtype::index row = threadIdx.x + blockIdx.x * blockDim.x;
    dtype::index column = (threadIdx.y + blockIdx.y * blockDim.y) * 2 * Matrix<dtype::real>::blockSize;

    // load vector to shared memory
    __shared__ dtype::real work[2 * Matrix<dtype::real>::blockSize * Matrix<dtype::real>::blockSize];
    work[threadIdx.x + threadIdx.y * 2 * Matrix<dtype::real>::blockSize] = column + threadIdx.x < rows ?
        vector[column + threadIdx.x] : 0.0f;
    __syncthreads();

    // compute partial vector product
    dtype::real product = 0.0f;
    for (dtype::index i = 0; i < 2 * Matrix<dtype::real>::blockSize; i++) {
        product += row < rows && column + i < rows ? matrix[row + (column + i) * rows] * work[i + threadIdx.y * 2 * Matrix<dtype::real>::blockSize] : 0.0f;
    }

    // set result
    if (row < rows) {
        result[row + (threadIdx.y + blockIdx.y * blockDim.y) * rows] = product;
    }
}

// row reduce kernel
__global__ void reduceRowKernel(dtype::real* vector, dtype::size rows) {
    // get id
    dtype::index row = blockIdx.x * blockDim.x + threadIdx.x;

    // check row
    if (row >= rows) {
        return;
    }

    // sum row
    dtype::real sum = 0.0f;
    dtype::size count = (rows + 2 * Matrix<dtype::real>::blockSize - 1) / (2 * Matrix<dtype::real>::blockSize);
    for (dtype::index i = 0; i < count; i++) {
        sum += vector[row + i * rows];
    }

    // set sum
    vector[row] = sum;
}

// fast gemv
void Conjugate::gemv(Matrix<dtype::real>& result, Matrix<dtype::real>& matrix,
    Matrix<dtype::real>& vector, hipStream_t stream) {
    // dimension
    dim3 blocks((matrix.rows() + 2 * Matrix<dtype::real>::blockSize - 1) / (2 * Matrix<dtype::real>::blockSize),
        (matrix.rows() / (2 * Matrix<dtype::real>::blockSize) + Matrix<dtype::real>::blockSize - 1) / Matrix<dtype::real>::blockSize);
    dim3 threads(2 * Matrix<dtype::real>::blockSize, Matrix<dtype::real>::blockSize);

    // call gemv kernel
    gemvKernel<<<blocks, threads, 0, stream>>>(matrix.deviceData(), vector.deviceData(),
        result.deviceData(), matrix.rows());

    // call reduce kernel
    reduceRowKernel<<<(matrix.columns() + Matrix<dtype::real>::blockSize * Matrix<dtype::real>::blockSize - 1) /
        (Matrix<dtype::real>::blockSize * Matrix<dtype::real>::blockSize),
        Matrix<dtype::real>::blockSize * Matrix<dtype::real>::blockSize, 0, stream>>>(result.deviceData(), result.rows());
}

