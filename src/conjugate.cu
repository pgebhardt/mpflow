#include "hip/hip_runtime.h"
// ert
//
// Copyright (C) 2012  Patrik Gebhardt
// Contact: patrik.gebhardt@rub.de
//
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.

// redefine extern c
#define LINALGCU_EXTERN_C extern "C"

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <cuda/hip/hip_runtime.h>
#include <cuda/hipblas.h>
#include <linalgcu/linalgcu.h>
#include "conjugate.h"

// add scalar kernel
__global__ void add_scalar_kernel(linalgcu_matrix_data_t* vector,
    linalgcu_matrix_data_t scalar) {
    // get id
    linalgcu_size_t i = blockIdx.x * blockDim.x + threadIdx.x;

    // add data
    vector[i] += scalar;
}

// add scalar
extern "C"
linalgcu_error_t ert_conjugate_add_scalar(linalgcu_matrix_t vector,
    linalgcu_matrix_data_t scalar) {
    // check input
    if (vector == NULL) {
        return LINALGCU_ERROR;
    }

    // execute kernel
    add_scalar_kernel<<<vector->size_m / LINALGCU_BLOCK_SIZE, LINALGCU_BLOCK_SIZE>>>(
        vector->device_data, scalar);

    return LINALGCU_SUCCESS;
}

