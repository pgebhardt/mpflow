#include "hip/hip_runtime.h"
// fastEIT
//
// Copyright (C) 2012  Patrik Gebhardt
// Contact: patrik.gebhardt@rub.de

#include "../include/fasteit.hpp"

// namespaces
using namespace fastEIT;
using namespace std;

// calc jacobian kernel
template<class BasisFunction>
__global__ void calc_jacobian_kernel(linalgcuMatrixData_t* jacobian,
    linalgcuMatrixData_t* drivePhi,
    linalgcuMatrixData_t* measurmentPhi,
    linalgcuMatrixData_t* connectivityMatrix,
    linalgcuMatrixData_t* elementalJacobianMatrix,
    linalgcuMatrixData_t* gamma, linalgcuMatrixData_t sigmaRef,
    linalgcuSize_t rows, linalgcuSize_t columns,
    linalgcuSize_t phiRows, linalgcuSize_t elementCount,
    linalgcuSize_t driveCount, linalgcuSize_t measurmentCount, bool additiv) {
    // get id
    linalgcuSize_t row = blockIdx.x * blockDim.x + threadIdx.x;
    linalgcuSize_t column = blockIdx.y * blockDim.y + threadIdx.y;

    // check column
    if (column >= elementCount) {
        return;
    }

    // calc measurment and drive id
    linalgcuSize_t roundMeasurmentCount = ((measurmentCount + LINALGCU_BLOCK_SIZE - 1) /
        LINALGCU_BLOCK_SIZE) * LINALGCU_BLOCK_SIZE;
    linalgcuSize_t measurmentId = row % roundMeasurmentCount;
    linalgcuSize_t driveId = row / roundMeasurmentCount;

    // variables
    linalgcuMatrixData_t dPhi[BasisFunction::nodesPerElement], mPhi[BasisFunction::nodesPerElement];
    linalgcuMatrixData_t id;

    // get data
    for (int i = 0; i < BasisFunction::nodesPerElement; i++) {
        id = connectivityMatrix[column + i * columns];
        dPhi[i] = driveId < driveCount ? drivePhi[(linalgcuSize_t)id + driveId * phiRows] : 0.0f;
        mPhi[i] = measurmentId < measurmentCount ? measurmentPhi[(linalgcuSize_t)id +
            measurmentId * phiRows] : 0.0f;
    }

    // calc matrix element
    linalgcuMatrixData_t element = 0.0f;
    for (int i = 0; i < BasisFunction::nodesPerElement; i++) {
        for (int j = 0; j < BasisFunction::nodesPerElement; j++) {
            element += dPhi[i] * mPhi[j] * elementalJacobianMatrix[column +
                (i + j * BasisFunction::nodesPerElement) * columns];
        }
    }

    // diff sigma to gamma
    element *= sigmaRef * exp10f(gamma[column] / 10.0f) / 10.0f;

    // set matrix element
    if (additiv == true) {
        jacobian[row + column * rows] += -element;
    }
    else {
        jacobian[row + column * rows] = -element;
    }
}

// calc jacobian
template
<
    class BasisFunction,
    class NumericSolver
>
linalgcuMatrix_t ForwardSolver<BasisFunction, NumericSolver>::calc_jacobian(linalgcuMatrix_t gamma,
    linalgcuSize_t harmonic, bool additiv, hipStream_t stream) const {
    // check input
    if (gamma == NULL) {
        throw invalid_argument("ForwardSolver::calc_jacobian: gamma == NULL");
    }
    if (harmonic > this->model()->numHarmonics()) {
        throw invalid_argument("ForwardSolver::calc_jacobian: harmonic > this->model()->numHarmonics()");
    }

    // dimension
    dim3 blocks(this->jacobian()->rows / LINALGCU_BLOCK_SIZE,
        this->jacobian()->columns / LINALGCU_BLOCK_SIZE);
    dim3 threads(LINALGCU_BLOCK_SIZE, LINALGCU_BLOCK_SIZE);

    // calc jacobian
    calc_jacobian_kernel<BasisFunction><<<blocks, threads, 0, stream>>>(
        this->jacobian()->deviceData, this->phi(harmonic)->deviceData,
        &this->phi(harmonic)->deviceData[this->driveCount() * this->phi(harmonic)->rows],
        this->model()->mesh()->elements()->deviceData, this->mElementalJacobianMatrix->deviceData,
        gamma->deviceData, this->model()->sigmaRef(), this->jacobian()->rows, this->jacobian()->columns,
        this->phi(harmonic)->rows, this->model()->mesh()->elementCount(),
        this->driveCount(), this->measurmentCount(), additiv);

    return LINALGCU_SUCCESS;
}

// specialisation
template class ForwardSolver<LinearBasis, SparseConjugate>;
