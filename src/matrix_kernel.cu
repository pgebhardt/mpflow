#include "hip/hip_runtime.h"
// fastEIT
//
// Copyright (C) 2012  Patrik Gebhardt
// Contact: patrik.gebhardt@rub.de

#include <cstdlib>
#include <cstdio>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "fasteit/cuda_error.h"

#include "fasteit/dtype.h"
#include "fasteit/constants.h"
#include "fasteit/matrix_kernel.h"


// add kernel
template <
    class type
>
__global__ void addKernel(const type* matrix, fastEIT::dtype::size rows, type* result) {
    // get ids
    fastEIT::dtype::index row = blockIdx.x * blockDim.x + threadIdx.x;
    fastEIT::dtype::index column = blockIdx.y * blockDim.y + threadIdx.y;

    // add B to A
    result[row + column * rows] += matrix[row + column * rows];
}

// add kernel wrapper
template <
    class type
>
void fastEIT::matrixKernel::add(dim3 blocks, dim3 threads, hipStream_t stream,
    const type* matrix, fastEIT::dtype::size rows, type* result) {
    // call cuda kernel
    addKernel<type><<<blocks, threads, 0, stream>>>(matrix, rows, result);

    CudaCheckError();
}

// add specialisation
template void fastEIT::matrixKernel::add<fastEIT::dtype::real>(
    dim3, dim3, hipStream_t, const fastEIT::dtype::real*,
    fastEIT::dtype::size, fastEIT::dtype::real*);
template void fastEIT::matrixKernel::add<fastEIT::dtype::index>(
    dim3, dim3, hipStream_t, const fastEIT::dtype::index*,
    fastEIT::dtype::size, fastEIT::dtype::index*);

// scale kernel
template <
    class type
>
__global__ void scaleKernel(type scalar, fastEIT::dtype::size rows, type* result) {
    // get ids
    fastEIT::dtype::index row = blockIdx.x * blockDim.x + threadIdx.x;
    fastEIT::dtype::index column = blockIdx.y * blockDim.y + threadIdx.y;

    // scale matrix with scalar
    result[row + column * rows] *= scalar;
}

// scale kernel wrapper
template <
    class type
>
void fastEIT::matrixKernel::scale(dim3 blocks, dim3 threads, hipStream_t stream,
    type scalar, dtype::size rows, type* result) {
    // call cuda kernel
    scaleKernel<type><<<blocks, threads, 0, stream>>>(scalar, rows, result);

    CudaCheckError();
}

// scale specialisation
template void fastEIT::matrixKernel::scale<fastEIT::dtype::real>(
    dim3, dim3, hipStream_t, fastEIT::dtype::real, fastEIT::dtype::size,
    fastEIT::dtype::real*);
template void fastEIT::matrixKernel::scale<fastEIT::dtype::index>(
    dim3, dim3, hipStream_t, fastEIT::dtype::index, fastEIT::dtype::size,
    fastEIT::dtype::index*);

// vector dot product kernel
template <
    class type
>
__global__ void vectorDotProductKernel(const type* a, const type* b, fastEIT::dtype::size rows,
    type* result) {
    // get ids
    fastEIT::dtype::index row = blockIdx.x * blockDim.x + threadIdx.x;
    fastEIT::dtype::index column = blockIdx.y * blockDim.y + threadIdx.y;

    // elementwise multiply
    result[row + column * rows] = a[row + column * rows] * b[row + column * rows];
}

// vector dot product kernel wrapper
template <
    class type
>
void fastEIT::matrixKernel::vectorDotProduct(dim3 blocks, dim3 threads,
    hipStream_t stream, const type* a, const type* b, dtype::size rows,
    type* result) {
    // call cuda kernel
    vectorDotProductKernel<type><<<blocks, threads, 0, stream>>>(
        a, b, rows, result);

    CudaCheckError();
}

// vector dot product specialisation
template void fastEIT::matrixKernel::vectorDotProduct<fastEIT::dtype::real>(
    dim3, dim3, hipStream_t, const fastEIT::dtype::real*,
    const fastEIT::dtype::real*, fastEIT::dtype::size, fastEIT::dtype::real*);
template void fastEIT::matrixKernel::vectorDotProduct<fastEIT::dtype::index>(
    dim3, dim3, hipStream_t, const fastEIT::dtype::index*,
    const fastEIT::dtype::index*, fastEIT::dtype::size, fastEIT::dtype::index*);

// sum kernel
template <
    class type
>
__global__ void sumKernel(const type* vector, fastEIT::dtype::size rows, fastEIT::dtype::size offset,
    type* result) {
    // get column
    fastEIT::dtype::index column = blockIdx.y * blockDim.y + threadIdx.y;

    // get id
    fastEIT::dtype::index gid = blockIdx.x * blockDim.x + threadIdx.x;
    fastEIT::dtype::index lid = threadIdx.x;

    // copy data to shared memory
    __volatile __shared__ type res[fastEIT::matrix::block_size * fastEIT::matrix::block_size];
    res[lid + threadIdx.y * fastEIT::matrix::block_size] =
        gid * offset < rows ? vector[gid * offset + column * rows] : 0.0f;

    // reduce
    res[lid + threadIdx.y * fastEIT::matrix::block_size] +=
        (lid % 2 == 0) ? res[lid + 1 + threadIdx.y * fastEIT::matrix::block_size] : 0.0f;
    res[lid + threadIdx.y * fastEIT::matrix::block_size] +=
        (lid % 4 == 0) ? res[lid + 2 + threadIdx.y * fastEIT::matrix::block_size] : 0.0f;
    res[lid + threadIdx.y * fastEIT::matrix::block_size] +=
        (lid % 8 == 0) ? res[lid + 4 + threadIdx.y * fastEIT::matrix::block_size] : 0.0f;
    res[lid + threadIdx.y * fastEIT::matrix::block_size] +=
        (lid % 16 == 0) ? res[lid + 8 + threadIdx.y * fastEIT::matrix::block_size] : 0.0f;
    __syncthreads();

    // stop rest of worker
    if (lid != 0) {
        return;
    }

    // write to global memory
    result[gid * offset + column * rows] = res[lid + threadIdx.y * fastEIT::matrix::block_size];
}

// sum kernel wrapper
template <
    class type
>
void fastEIT::matrixKernel::sum(dim3 blocks, dim3 threads, hipStream_t stream,
    const type* vector, dtype::size rows, dtype::size offset, type* result) {
    // call cuda kernel
    sumKernel<type><<<blocks, threads, 0, stream>>>(vector, rows, offset, result);

    CudaCheckError();
}

// sum specialisation
template void fastEIT::matrixKernel::sum<fastEIT::dtype::real>(dim3, dim3,
    hipStream_t, const fastEIT::dtype::real*, fastEIT::dtype::size,
    fastEIT::dtype::size, fastEIT::dtype::real*);
template void fastEIT::matrixKernel::sum<fastEIT::dtype::index>(dim3, dim3,
    hipStream_t, const fastEIT::dtype::index*, fastEIT::dtype::size,
    fastEIT::dtype::size, fastEIT::dtype::index*);

// min kernel
template <
    class type
>
__global__ void minKernel(const type* vector, fastEIT::dtype::size rows, fastEIT::dtype::size offset, type* result) {
    // get id
    fastEIT::dtype::index gid = blockIdx.x * blockDim.x + threadIdx.x;
    fastEIT::dtype::index lid = threadIdx.x;

    // copy data to shared memory
    __volatile __shared__ type res[fastEIT::matrix::block_size];
    res[lid] = gid * offset < rows ? vector[gid * offset] : NAN;

    // reduce
    res[lid] = (lid % 2 == 0) ? (res[lid + 1] < res[lid] ? res[lid + 1] : res[lid]) : res[lid];
    res[lid] = (lid % 4 == 0) ? (res[lid + 2] < res[lid] ? res[lid + 2] : res[lid]) : res[lid];
    res[lid] = (lid % 8 == 0) ? (res[lid + 4] < res[lid] ? res[lid + 4] : res[lid]) : res[lid];
    res[lid] = (lid % 16 == 0) ? (res[lid + 8] < res[lid] ? res[lid + 8] : res[lid]) : res[lid];

    // stop rest of worker
    if (lid != 0) {
        return;
    }

    // write to global memory
    result[blockIdx.x * blockDim.x * offset] = res[0];
}

// min kernel wrapper
template <
    class type
>
void fastEIT::matrixKernel::min(dim3 blocks, dim3 threads, hipStream_t stream,
    const type* vector, dtype::size rows, dtype::size offset, type* result) {
    // call cuda kernel
    minKernel<type><<<blocks, threads, 0, stream>>>(vector, rows, offset, result);

    CudaCheckError();
}

// min specialisation
template void fastEIT::matrixKernel::min<fastEIT::dtype::real>(dim3, dim3,
    hipStream_t, const fastEIT::dtype::real*, fastEIT::dtype::size,
    fastEIT::dtype::size, fastEIT::dtype::real*);
template void fastEIT::matrixKernel::min<fastEIT::dtype::index>(dim3, dim3,
    hipStream_t, const fastEIT::dtype::index*, fastEIT::dtype::size,
    fastEIT::dtype::size, fastEIT::dtype::index*);

// max kernel
template <
    class type
>
__global__ void maxKernel(const type* vector, fastEIT::dtype::size rows, fastEIT::dtype::size offset, type* result) {
    // get id
    fastEIT::dtype::index gid = blockIdx.x * blockDim.x + threadIdx.x;
    fastEIT::dtype::index lid = threadIdx.x;

    // copy data to shared memory
    __volatile __shared__ type res[fastEIT::matrix::block_size];
    res[lid] = gid * offset < rows ? vector[gid * offset] : NAN;

    // reduce
    res[lid] = (lid % 2 == 0) ? (res[lid + 1] > res[lid] ? res[lid + 1] : res[lid]) : res[lid];
    res[lid] = (lid % 4 == 0) ? (res[lid + 2] > res[lid] ? res[lid + 2] : res[lid]) : res[lid];
    res[lid] = (lid % 8 == 0) ? (res[lid + 4] > res[lid] ? res[lid + 4] : res[lid]) : res[lid];
    res[lid] = (lid % 16 == 0) ? (res[lid + 8] > res[lid] ? res[lid + 8] : res[lid]) : res[lid];

    // stop rest of worker
    if (lid != 0) {
        return;
    }

    // write to global memory
    result[blockIdx.x * blockDim.x * offset] = res[0];
}

// max kernel wrapper
template <
    class type
>
void fastEIT::matrixKernel::max(dim3 blocks, dim3 threads, hipStream_t stream,
    const type* vector, dtype::size rows, dtype::size offset, type* result) {
    // call cuda kernel
    maxKernel<type><<<blocks, threads, 0, stream>>>(vector, rows, offset, result);

    CudaCheckError();
}

// max specialisation
template void fastEIT::matrixKernel::max<fastEIT::dtype::real>(dim3, dim3,
    hipStream_t, const fastEIT::dtype::real*, fastEIT::dtype::size,
    fastEIT::dtype::size, fastEIT::dtype::real*);
template void fastEIT::matrixKernel::max<fastEIT::dtype::index>(dim3, dim3,
    hipStream_t, const fastEIT::dtype::index*, fastEIT::dtype::size,
    fastEIT::dtype::size, fastEIT::dtype::index*);
