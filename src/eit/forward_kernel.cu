#include "hip/hip_runtime.h"
// mpFlow
//
// Copyright (C) 2013  Patrik Gebhardt
// Contact: patrik.gebhardt@rub.de

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "mpflow/cuda_error.h"

#include "mpflow/dtype.h"
#include "mpflow/numeric/constants.h"
#include "mpflow/eit/forward_kernel.h"

// calc voltage kernel
static __global__ void applyMeasurementPatternKernel(const mpFlow::dtype::real* potential,
    mpFlow::dtype::size offset, mpFlow::dtype::size rows, const mpFlow::dtype::real* pattern,
    mpFlow::dtype::size pattern_rows, bool additiv,
    mpFlow::dtype::real* voltage, mpFlow::dtype::size voltage_rows) {
    // get ids
    mpFlow::dtype::index row = blockIdx.x * blockDim.x + threadIdx.x;
    mpFlow::dtype::index column = blockIdx.y * blockDim.y + threadIdx.y;

    // calc voltage
    mpFlow::dtype::real value = 0.0f;
    for (mpFlow::dtype::index electrode = 0; electrode < pattern_rows; ++electrode) {
        value += pattern[electrode + pattern_rows * row] * potential[offset + electrode + column * rows];
    }

    // set voltage
    if (additiv == true) {
        voltage[row + voltage_rows * column] += value;
    } else {
        voltage[row + voltage_rows * column] = value;
    }
}

// calc voltage kernel wrapper
void mpFlow::EIT::forwardKernel::applyMeasurementPattern(dim3 blocks, dim3 threads, hipStream_t stream,
    const mpFlow::dtype::real* potential, mpFlow::dtype::size offset,
    mpFlow::dtype::size rows, const mpFlow::dtype::real* pattern,
    mpFlow::dtype::size pattern_rows, bool additiv,
    mpFlow::dtype::real* voltage, mpFlow::dtype::size voltage_rows) {
    // call cuda kernel
    applyMeasurementPatternKernel<<<blocks, threads, 0, stream>>>(
        potential, offset, rows, pattern, pattern_rows, additiv, voltage, voltage_rows);

    CudaCheckError();
}

