#include "hip/hip_runtime.h"
// fastECT
//
// Copyright (C) 2012  Patrik Gebhardt
// Contact: patrik.gebhardt@rub.de

// redefine extern c
#define LINALGCU_EXTERN_C extern "C"

#include <stdlib.h>
#include "../include/fastect.h"

// calc residual integral
linalgcuMatrixData_t calc_residual_integral(
    linalgcuMatrixData_t x1, linalgcuMatrixData_t y1,
    linalgcuMatrixData_t x2, linalgcuMatrixData_t y2,
    linalgcuMatrixData_t x3, linalgcuMatrixData_t y3,
    linalgcuMatrixData_t ai, linalgcuMatrixData_t bi, linalgcuMatrixData_t ci,
    linalgcuMatrixData_t aj, linalgcuMatrixData_t bj, linalgcuMatrixData_t cj) {
    // calc area
    linalgcuMatrixData_t area = 0.5 * fabs((x2 - x1) * (y3 - y1) -
        (x3 - x1) * (y2 - y1));

    // calc integral
    linalgcuMatrixData_t integral = 2.0f * area *
        (ai * (0.5f * aj + (1.0f / 6.0f) * bj * (x1 + x2 + x3) +
        (1.0f / 6.0f) * cj * (y1 + y2 + y3)) +
        bi * ((1.0f/ 6.0f) * aj * (x1 + x2 + x3) +
        (1.0f / 12.0f) * bj * (x1 * x1 + x1 * x2 + x1 * x3 + x2 * x2 + x2 * x3 + x3 * x3) +
        (1.0f/ 24.0f) * cj * (2.0f * x1 * y1 + x1 * y2 + x1 * y3 + x2 * y1 +
        2.0f * x2 * y2 + x2 * y3 + x3 * y1 + x3 * y2 + 2.0f * x3 * y3)) +
        ci * ((1.0f / 6.0f) * aj * (y1 + y2 + y3) +
        (1.0f / 12.0f) * cj * (y1 * y1 + y1 * y2 + y1 * y3 + y2 * y2 + y2 * y3 + y3 * y3) +
        (1.0f / 24.0f) * bj * (2.0f * x1 * y1 + x1 * y2 + x1 * y3 + x2 * y1 +
        2.0f * x2 * y2 + x2 * y3 + x3 * y1 + x3 * y2 + 2.0f * x3 * y3)));

    return integral;
}

// reduce connectivity and elementalResidual matrix
__global__ void reduce_residual_matrices(linalgcuMatrixData_t* connectivityMatrix,
    linalgcuMatrixData_t* elementalResidualMatrix,
    linalgcuMatrixData_t* intermediateConnectivityMatrix,
    linalgcuMatrixData_t* intermediateElementalResidualMatrix,
    linalgcuColumnId_t* systemMatrixColumnIds, linalgcuSize_t rows,
    linalgcuSize_t columns) {
    // get ids
    linalgcuSize_t row = blockIdx.x * blockDim.x + threadIdx.x;
    linalgcuSize_t column = blockIdx.y * blockDim.y + threadIdx.y;

    // get column id
    linalgcuColumnId_t columnId = systemMatrixColumnIds[row * LINALGCU_BLOCK_SIZE + column];

    // check column id
    if (columnId == -1) {
        return;
    }

    // reduce matrices
    for (int k = 0; k < LINALGCU_BLOCK_SIZE; k++) {
        connectivityMatrix[row + (column + k * LINALGCU_BLOCK_SIZE) * rows] =
            intermediateConnectivityMatrix[row + (columnId + k * columns) * rows];

        elementalResidualMatrix[row + (column + k * LINALGCU_BLOCK_SIZE) * rows] =
            intermediateElementalResidualMatrix[row + (columnId + k * columns) * rows];
    }
}

// init residual matrix
LINALGCU_EXTERN_C
linalgcuError_t fastect_grid_init_residual_matrix(fastectGrid_t grid,
    linalgcuMatrix_t gamma, hipStream_t stream) {
    // check input
    if ((grid == NULL) || (gamma == NULL)) {
        return LINALGCU_ERROR;
    }

    // error
    linalgcuError_t error = LINALGCU_SUCCESS;

    // create intermediate matrices
    linalgcuMatrix_t elementCount, connectivityMatrix, elementalResidualMatrix;
    error  = linalgcu_matrix_create(&elementCount, grid->mesh->vertexCount,
        grid->mesh->vertexCount, stream);
    error |= linalgcu_matrix_create(&connectivityMatrix, grid->connectivityMatrix->rows,
        elementCount->columns * LINALGCU_BLOCK_SIZE, stream);
    error |= linalgcu_matrix_create(&elementalResidualMatrix,
        grid->elementalResidualMatrix->rows, elementCount->columns * LINALGCU_BLOCK_SIZE,
        stream);

    // check success
    if (error != LINALGCU_SUCCESS) {
        return error;
    }

    // init connectivityMatrix
    for (linalgcuSize_t i = 0; i < grid->connectivityMatrix->rows; i++) {
        for (linalgcuSize_t j = 0; j < grid->connectivityMatrix->columns; j++) {
            linalgcu_matrix_set_element(grid->connectivityMatrix, -1.0f, i, j);
        }
    }
    linalgcu_matrix_copy_to_device(grid->connectivityMatrix, stream);

    // fill intermediate connectivity and elementalResidual matrices
    linalgcuMatrixData_t id[3], x[3], y[3];
    linalgcuMatrixData_t temp;
    fastectBasis_t basis[3];

    for (linalgcuSize_t k = 0; k < grid->mesh->elementCount; k++) {
        // get vertices for element
        for (linalgcuSize_t i = 0; i < 3; i++) {
            linalgcu_matrix_get_element(grid->mesh->elements, &id[i], k, i);
            linalgcu_matrix_get_element(grid->mesh->vertices, &x[i],
                (linalgcuSize_t)id[i], 0);
            linalgcu_matrix_get_element(grid->mesh->vertices, &y[i],
                (linalgcuSize_t)id[i], 1);
        }

        // calc corresponding basis functions
        fastect_basis_create(&basis[0], x[0], y[0], x[1], y[1], x[2], y[2]);
        fastect_basis_create(&basis[1], x[1], y[1], x[2], y[2], x[0], y[0]);
        fastect_basis_create(&basis[2], x[2], y[2], x[0], y[0], x[1], y[1]);

        // set connectivity and elemental residual matrix elements
        for (linalgcuSize_t i = 0; i < 3; i++) {
            for (linalgcuSize_t j = 0; j < 3; j++) {
                // get current element count
                linalgcu_matrix_get_element(elementCount, &temp,
                    (linalgcuSize_t)id[i], (linalgcuSize_t)id[j]);

                // set connectivity element
                linalgcu_matrix_set_element(connectivityMatrix,
                    (linalgcuMatrixData_t)k, (linalgcuSize_t)id[i],
                    (linalgcuSize_t)(id[j] + connectivityMatrix->rows * temp));

                // set elemental residual element
                linalgcu_matrix_set_element(elementalResidualMatrix,
                    calc_residual_integral(x[0], y[0], x[1], y[1], x[2], y[2],
                        basis[i]->coefficients[0], basis[i]->coefficients[1],
                        basis[i]->coefficients[2], basis[j]->coefficients[0],
                        basis[j]->coefficients[1], basis[j]->coefficients[2]),
                    (linalgcuSize_t)id[i],
                    (linalgcuSize_t)(id[j] + connectivityMatrix->rows * temp));

                // increment element count
                elementCount->hostData[(linalgcuSize_t)id[i] + (linalgcuSize_t)id[j] *
                    elementCount->rows] += 1.0f;
            }
        }

        // cleanup
        fastect_basis_release(&basis[0]);
        fastect_basis_release(&basis[1]);
        fastect_basis_release(&basis[2]);
    }

    // upload intermediate matrices
    linalgcu_matrix_copy_to_device(connectivityMatrix, stream);
    linalgcu_matrix_copy_to_device(elementalResidualMatrix, stream);

    // reduce matrices
    dim3 blocks(connectivityMatrix->rows / LINALGCU_BLOCK_SIZE, 1);
    dim3 threads(LINALGCU_BLOCK_SIZE, LINALGCU_BLOCK_SIZE);
    reduce_residual_matrices<<<blocks, threads, 0, stream>>>(
        grid->connectivityMatrix->deviceData,
        grid->elementalResidualMatrix->deviceData,
        connectivityMatrix->deviceData,
        elementalResidualMatrix->deviceData,
        grid->systemMatrix2D->columnIds,
        grid->connectivityMatrix->rows, grid->connectivityMatrix->rows);

    // update residual matrix
    error = fastect_grid_update_residual_matrix(grid, gamma, stream);

    // cleanup
    linalgcu_matrix_release(&elementCount);
    linalgcu_matrix_release(&connectivityMatrix);
    linalgcu_matrix_release(&elementalResidualMatrix);

    return LINALGCU_SUCCESS;
}

// update_system_matrix_kernel
__global__ void update_system_matrix_kernel(linalgcuMatrixData_t* systemMatrixValues,
    linalgcuColumnId_t* systemMatrixColumnIds,
    linalgcuMatrixData_t* gradientMatrixTransposedValues,
    linalgcuColumnId_t* gradientMatrixTransposedColumnIds,
    linalgcuMatrixData_t* gradientMatrixTransposed,
    linalgcuMatrixData_t* gamma, linalgcuMatrixData_t sigmaRef,
    linalgcuMatrixData_t* area,
    linalgcuSize_t gradientMatrixTransposedRows) {
    // get ids
    linalgcuSize_t i = blockIdx.x * blockDim.x + threadIdx.x;
    linalgcuColumnId_t j = systemMatrixColumnIds[i * LINALGCU_BLOCK_SIZE +
        (blockIdx.y * blockDim.y + threadIdx.y)];

    // calc system matrix elements
    linalgcuMatrixData_t element = 0.0f;
    linalgcuColumnId_t id = -1;

    for (int k = 0; k < LINALGCU_BLOCK_SIZE; k++) {
        // get id
        id = gradientMatrixTransposedColumnIds[i * LINALGCU_BLOCK_SIZE + k];

        element += id != -1 && j != -1 ?
            gradientMatrixTransposedValues[i * LINALGCU_BLOCK_SIZE + k] *
            sigmaRef * exp10f(gamma[id / 2] / 10.0f) * area[id / 2] *
            gradientMatrixTransposed[j + id * gradientMatrixTransposedRows] :
            0.0f;
    }

    // set element
    systemMatrixValues[i * LINALGCU_BLOCK_SIZE + (blockIdx.y * blockDim.y + threadIdx.y)] =
        element;
}

// update system matrix 2D
LINALGCU_EXTERN_C
linalgcuError_t fastect_grid_update_2D_system_matrix(fastectGrid_t grid,
    linalgcuMatrix_t gamma, hipStream_t stream) {
    // check input
    if ((grid == NULL) || (gamma == NULL)) {
        return LINALGCU_ERROR;
    }

    // dimension
    dim3 threads(LINALGCU_BLOCK_SIZE, LINALGCU_BLOCK_SIZE);
    dim3 blocks(grid->systemMatrix2D->rows / LINALGCU_BLOCK_SIZE, 1);

    // execute kernel
    update_system_matrix_kernel<<<blocks, threads, 0, stream>>>(
        grid->systemMatrix2D->values,
        grid->systemMatrix2D->columnIds,
        grid->gradientMatrixTransposedSparse->values,
        grid->gradientMatrixTransposedSparse->columnIds,
        grid->gradientMatrixTransposed->deviceData,
        gamma->deviceData, grid->sigmaRef,
        grid->area->deviceData,
        grid->gradientMatrixTransposed->rows);

    return LINALGCU_SUCCESS;
}

// update residual matrix kernel
__global__ void update_residual_matrix_kernel(linalgcuMatrixData_t* residualMatrixValues,
    linalgcuColumnId_t* residualMatrixColumnIds,
    linalgcuColumnId_t* systemMatrixColumnIds,
    linalgcuMatrixData_t* connectivityMatrix,
    linalgcuMatrixData_t* elementalResidualMatrix,
    linalgcuMatrixData_t* gamma, linalgcuMatrixData_t sigmaRef,
    linalgcuSize_t rows) {
    // get ids
    linalgcuSize_t row = blockIdx.x * blockDim.x + threadIdx.x;
    linalgcuSize_t column = blockIdx.y * blockDim.y + threadIdx.y;

    // get columnId
    linalgcuColumnId_t columnId = systemMatrixColumnIds[row * LINALGCU_BLOCK_SIZE + column];

    // set column id
    residualMatrixColumnIds[row * LINALGCU_BLOCK_SIZE + column] = columnId;

    // check column id
    if (columnId == -1) {
        return;
    }

    // calc residual matrix element
    linalgcuMatrixData_t value = 0.0f;
    linalgcuColumnId_t elementId = -1;
    for (int k = 0; k < LINALGCU_BLOCK_SIZE; k++) {
        // get element id
        elementId = (linalgcuColumnId_t)connectivityMatrix[row +
            (column + k * LINALGCU_BLOCK_SIZE) * rows];

        value += elementId != -1 ? elementalResidualMatrix[row +
            (column + k * LINALGCU_BLOCK_SIZE) * rows] *
            sigmaRef * exp10f(gamma[elementId] / 10.0f) : 0.0f;
    }

    // set residual matrix element
    residualMatrixValues[row * LINALGCU_BLOCK_SIZE + column] = value;
}

// update residual matrix
LINALGCU_EXTERN_C
linalgcuError_t fastect_grid_update_residual_matrix(fastectGrid_t grid,
    linalgcuMatrix_t gamma, hipStream_t stream) {
    // check input
    if ((grid == NULL) || (gamma == NULL)) {
        return LINALGCU_ERROR;
    }

    // dimension
    dim3 threads(LINALGCU_BLOCK_SIZE, LINALGCU_BLOCK_SIZE);
    dim3 blocks(grid->residualMatrix->rows / LINALGCU_BLOCK_SIZE, 1);

    // execute kernel
    update_residual_matrix_kernel<<<blocks, threads, 0, stream>>>(
        grid->residualMatrix->values, grid->residualMatrix->columnIds,
        grid->systemMatrix2D->columnIds, grid->connectivityMatrix->deviceData,
        grid->elementalResidualMatrix->deviceData, gamma->deviceData,
        grid->sigmaRef, grid->connectivityMatrix->rows);

    return LINALGCU_SUCCESS;
}
