#include "hip/hip_runtime.h"
// --------------------------------------------------------------------
// This file is part of mpFlow.
//
// mpFlow is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 2 of the License, or
// (at your option) any later version.
//
// mpFlow is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with mpFlow. If not, see <http://www.gnu.org/licenses/>.
//
// Copyright (C) 2014 Patrik Gebhardt
// Contact: patrik.gebhardt@rub.de
// --------------------------------------------------------------------

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <thrust/complex.h>
#include "mpflow/cuda_error.h"

#include "mpflow/dtype.h"
#include "mpflow/numeric/constants.h"
#include "mpflow/numeric/bicgstab_kernel.h"

// update vector kernel
template <
    class dataType
>
static __global__ void updateVectorKernel(const dataType* x1,
    const mpFlow::dtype::real sign, const dataType* x2, const dataType* scalar,
    mpFlow::dtype::size rows, dataType* result) {
    // get ids
    mpFlow::dtype::index row = blockIdx.x * blockDim.x + threadIdx.x;
    mpFlow::dtype::index column = blockIdx.y * blockDim.y + threadIdx.y;

    // calc value
    result[row + column * rows] = x1[row + column * rows] +
        sign * scalar[column * rows] * x2[row + column * rows];
}

// update vector kernel wrapper
template <
    class dataType
>
void mpFlow::numeric::bicgstabKernel::updateVector(dim3 blocks, dim3 threads,
    hipStream_t stream, const dataType* x1, const dtype::real sign,
    const dataType* x2, const dataType* scalar, dtype::size rows,
    dataType* result) {
    // call cuda kernel
    updateVectorKernel<<<blocks, threads, 0, stream>>>(x1, sign, x2, scalar, rows,
        result);

    CudaCheckError();
}

template void mpFlow::numeric::bicgstabKernel::updateVector<mpFlow::dtype::real>(dim3, dim3,
    hipStream_t, const mpFlow::dtype::real*, const mpFlow::dtype::real,
    const mpFlow::dtype::real*, const mpFlow::dtype::real*, mpFlow::dtype::size,
    mpFlow::dtype::real*);
template void mpFlow::numeric::bicgstabKernel::updateVector<mpFlow::dtype::complex>(dim3, dim3,
    hipStream_t, const mpFlow::dtype::complex*, const mpFlow::dtype::real,
    const mpFlow::dtype::complex*, const mpFlow::dtype::complex*, mpFlow::dtype::size,
    mpFlow::dtype::complex*);
