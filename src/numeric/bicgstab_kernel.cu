#include "hip/hip_runtime.h"
// --------------------------------------------------------------------
// This file is part of mpFlow.
//
// mpFlow is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 2 of the License, or
// (at your option) any later version.
//
// mpFlow is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with mpFlow. If not, see <http://www.gnu.org/licenses/>.
//
// Copyright (C) 2014 Patrik Gebhardt
// Contact: patrik.gebhardt@rub.de
// --------------------------------------------------------------------

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <thrust/complex.h>
#include "mpflow/cuda_error.h"

#include "mpflow/dtype.h"
#include "mpflow/numeric/constants.h"
#include "mpflow/numeric/bicgstab_kernel.h"

// update vector kernel
static __global__ void updateVectorKernel(const mpFlow::dtype::real* x1,
    const mpFlow::dtype::real sign, const mpFlow::dtype::real* x2,
    const mpFlow::dtype::real* scalar, mpFlow::dtype::size rows,
    mpFlow::dtype::real* result) {
    // get ids
    mpFlow::dtype::index row = blockIdx.x * blockDim.x + threadIdx.x;
    mpFlow::dtype::index column = blockIdx.y * blockDim.y + threadIdx.y;

    // calc value
    result[row + column * rows] = x1[row + column * rows] +
        sign * scalar[column * rows] * x2[row + column * rows];
}

// update vector kernel wrapper
void mpFlow::numeric::bicgstabKernel::updateVector(dim3 blocks, dim3 threads,
    hipStream_t stream, const dtype::real* x1, const dtype::real sign,
    const dtype::real* x2, const dtype::real* scalar, dtype::size rows,
    dtype::real* result) {
    // call cuda kernel
    updateVectorKernel<<<blocks, threads, 0, stream>>>(x1, sign, x2, scalar, rows,
        result);

    CudaCheckError();
}
