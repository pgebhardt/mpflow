#include "hip/hip_runtime.h"
// --------------------------------------------------------------------
// This file is part of mpFlow.
//
// mpFlow is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 2 of the License, or
// (at your option) any later version.
//
// mpFlow is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with mpFlow. If not, see <http://www.gnu.org/licenses/>.
//
// Copyright (C) 2014 Patrik Gebhardt
// Contact: patrik.gebhardt@rub.de
// --------------------------------------------------------------------

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <thrust/complex.h>
#include "mpflow/cuda_error.h"

#include "mpflow/dtype.h"
#include "mpflow/numeric/constants.h"
#include "mpflow/numeric/sparse_matrix_kernel.h"

// convert to sparse matrix kernel
template <
    class type
>
static __global__ void convertKernel(const type* matrix,
    mpFlow::dtype::size rows, mpFlow::dtype::size columns,
    type* values, mpFlow::dtype::index* columnIds,
    mpFlow::dtype::index* elementCount) {
    // get id
    mpFlow::dtype::index i = blockIdx.x * blockDim.x + threadIdx.x;

    // element count
    mpFlow::dtype::size count = 0;

    // init values and columnIds
    for (mpFlow::dtype::index j = 0; j < mpFlow::numeric::sparseMatrix::block_size; j++) {
        values[i * mpFlow::numeric::sparseMatrix::block_size + j] = 0.0f;
        columnIds[i * mpFlow::numeric::sparseMatrix::block_size + j] = mpFlow::dtype::invalid_index;
    }

    // search non-zero elements
    type element = 0.0f;
    for (mpFlow::dtype::index j = 0; j < columns; j++) {
        // get element
        element = matrix[i + j * rows];

        // check for non-zero
        if (element != type(0)) {
            values[i * mpFlow::numeric::sparseMatrix::block_size + count] = element;
            columnIds[i * mpFlow::numeric::sparseMatrix::block_size + count] = j;

            // increment count
            count++;

            // check count
            if (count >= mpFlow::numeric::sparseMatrix::block_size) {
                break;
            }
        }
    }

    // save element count
    elementCount[i] = count;
}

// convert to sparse matrix kernel wrapper
template <
    class type
>
void mpFlow::numeric::sparseMatrixKernel::convert(dim3 blocks, dim3 threads, hipStream_t stream,
    const type* matrix, dtype::size rows, dtype::size columns,
    type* values, dtype::index* columnIds, dtype::index* elementCount) {
    // call cuda kernel
    convertKernel<type><<<blocks, threads, 0, stream>>>(matrix, rows, columns,
        values, columnIds, elementCount);

    CudaCheckError();
}

// convert to matrix kernel
template <
    class type
>
static __global__ void convertToMatrixKernel(const type* values,
    const mpFlow::dtype::index* column_ids, mpFlow::dtype::size density,
    mpFlow::dtype::size rows, type* matrix) {
    // get row id
    mpFlow::dtype::index row = blockIdx.x * blockDim.x + threadIdx.x;

    // expand sparse matrix
    mpFlow::dtype::index column_id = mpFlow::dtype::invalid_index;
    for (mpFlow::dtype::index column = 0; column < density; ++column) {
        // get column id
        column_id = column_ids[row * mpFlow::numeric::sparseMatrix::block_size + column];

        // set matrix value
        if (column_id != mpFlow::dtype::invalid_index) {
            matrix[row + column_id * rows] = values[
                row * mpFlow::numeric::sparseMatrix::block_size + column];
        }
    }
}

// convert to matrix kernel wrapper
template <
    class type
>
void mpFlow::numeric::sparseMatrixKernel::convertToMatrix(dim3 blocks, dim3 threads,
    hipStream_t stream, const type* values, const dtype::index* column_ids,
    dtype::size density, dtype::size rows, type* matrix) {
    // call cuda kernel
    convertToMatrixKernel<type><<<blocks, threads, 0, stream>>>(values, column_ids,
        density, rows, matrix);

    CudaCheckError();
}

// sparse matrix multiply kernel
template <
    class type
>
static __global__ void multiplyKernel(const type* values,
    const mpFlow::dtype::index* columnIds, const type* matrix,
    mpFlow::dtype::size result_rows, mpFlow::dtype::size matrix_rows,
    mpFlow::dtype::size columns, mpFlow::dtype::size density, type* result) {
    // get ids
    mpFlow::dtype::index row = blockIdx.x * blockDim.x + threadIdx.x;
    mpFlow::dtype::index column = blockIdx.y * blockDim.y + threadIdx.y;

    // calc result
    type res = 0.0f;
    mpFlow::dtype::index id = mpFlow::dtype::invalid_index;

    // read column ids to local memory
    __shared__ mpFlow::dtype::index columnId[
        mpFlow::numeric::sparseMatrix::block_size * mpFlow::numeric::sparseMatrix::block_size];
    __shared__ type value[
        mpFlow::numeric::sparseMatrix::block_size * mpFlow::numeric::sparseMatrix::block_size];

    columnId[threadIdx.x * mpFlow::numeric::sparseMatrix::block_size + threadIdx.y] = row < result_rows ?
        columnIds[row * mpFlow::numeric::sparseMatrix::block_size + threadIdx.y] : mpFlow::dtype::invalid_index;
    value[threadIdx.x * mpFlow::numeric::sparseMatrix::block_size + threadIdx.y] = row < result_rows ?
        values[row * mpFlow::numeric::sparseMatrix::block_size + threadIdx.y] : 0.0f;

    __syncthreads();

    // check ids
    if ((row >= result_rows) || (column >= columns)) {
        return;
    }

    // read matrix to local memory
    for (mpFlow::dtype::index j = 0; j < density; j++) {
        // get column id
        id = columnId[threadIdx.x * mpFlow::numeric::sparseMatrix::block_size + j];

         res += id != mpFlow::dtype::invalid_index ? matrix[id + column * matrix_rows] *
            value[threadIdx.x * mpFlow::numeric::sparseMatrix::block_size + j] : 0.0f;
    }

    // set result
    result[row + column * result_rows] = res;
}

template <>
__global__ void multiplyKernel(const thrust::complex<float>* values,
    const mpFlow::dtype::index* columnIds, const thrust::complex<float>* matrix,
    mpFlow::dtype::size result_rows, mpFlow::dtype::size matrix_rows,
    mpFlow::dtype::size columns, mpFlow::dtype::size density, thrust::complex<float>* result) {
    // get ids
    mpFlow::dtype::index row = blockIdx.x * blockDim.x + threadIdx.x;
    mpFlow::dtype::index column = blockIdx.y * blockDim.y + threadIdx.y;

    // calc result
    hipFloatComplex res = make_hipFloatComplex(0.0f, 0.0f);
    mpFlow::dtype::index id = mpFlow::dtype::invalid_index;

    // read column ids to local memory
    __shared__ mpFlow::dtype::index columnId[
        mpFlow::numeric::sparseMatrix::block_size * mpFlow::numeric::sparseMatrix::block_size];
    __shared__ hipFloatComplex value[
        mpFlow::numeric::sparseMatrix::block_size * mpFlow::numeric::sparseMatrix::block_size];

    columnId[threadIdx.x * mpFlow::numeric::sparseMatrix::block_size + threadIdx.y] = row < result_rows ?
        columnIds[row * mpFlow::numeric::sparseMatrix::block_size + threadIdx.y] : mpFlow::dtype::invalid_index;
    value[threadIdx.x * mpFlow::numeric::sparseMatrix::block_size + threadIdx.y].x = row < result_rows ?
        values[row * mpFlow::numeric::sparseMatrix::block_size + threadIdx.y].real() : 0.0f;
    value[threadIdx.x * mpFlow::numeric::sparseMatrix::block_size + threadIdx.y].y = row < result_rows ?
        values[row * mpFlow::numeric::sparseMatrix::block_size + threadIdx.y].imag() : 0.0f;

    __syncthreads();

    // check ids
    if ((row >= result_rows) || (column >= columns)) {
        return;
    }

    // read matrix to local memory
    for (mpFlow::dtype::index j = 0; j < density; j++) {
        // get column id
        id = columnId[threadIdx.x * mpFlow::numeric::sparseMatrix::block_size + j];
        hipFloatComplex element = *(hipFloatComplex*)&matrix[id + column * matrix_rows];
        hipFloatComplex temp = id != mpFlow::dtype::invalid_index ? hipCmulf(element,
            value[threadIdx.x * mpFlow::numeric::sparseMatrix::block_size + j]) : make_hipFloatComplex(0.0f, 0.0f);

        res.x += temp.x;
        res.y += temp.y;
    }

    // set result
    result[row + column * result_rows].real(res.x);
    result[row + column * result_rows].imag(res.y);
}

template <>
__global__ void multiplyKernel(const thrust::complex<double>* values,
    const mpFlow::dtype::index* columnIds, const thrust::complex<double>* matrix,
    mpFlow::dtype::size result_rows, mpFlow::dtype::size matrix_rows,
    mpFlow::dtype::size columns, mpFlow::dtype::size density, thrust::complex<double>* result) {
    // get ids
    mpFlow::dtype::index row = blockIdx.x * blockDim.x + threadIdx.x;
    mpFlow::dtype::index column = blockIdx.y * blockDim.y + threadIdx.y;

    // calc result
    hipDoubleComplex res = make_hipDoubleComplex(0.0f, 0.0f);
    mpFlow::dtype::index id = mpFlow::dtype::invalid_index;

    // read column ids to local memory
    __shared__ mpFlow::dtype::index columnId[
        mpFlow::numeric::sparseMatrix::block_size * mpFlow::numeric::sparseMatrix::block_size];
    __shared__ hipDoubleComplex value[
        mpFlow::numeric::sparseMatrix::block_size * mpFlow::numeric::sparseMatrix::block_size];

    columnId[threadIdx.x * mpFlow::numeric::sparseMatrix::block_size + threadIdx.y] = row < result_rows ?
        columnIds[row * mpFlow::numeric::sparseMatrix::block_size + threadIdx.y] : mpFlow::dtype::invalid_index;
    value[threadIdx.x * mpFlow::numeric::sparseMatrix::block_size + threadIdx.y].x = row < result_rows ?
        values[row * mpFlow::numeric::sparseMatrix::block_size + threadIdx.y].real() : 0.0f;
    value[threadIdx.x * mpFlow::numeric::sparseMatrix::block_size + threadIdx.y].y = row < result_rows ?
        values[row * mpFlow::numeric::sparseMatrix::block_size + threadIdx.y].imag() : 0.0f;

    __syncthreads();

    // check ids
    if ((row >= result_rows) || (column >= columns)) {
        return;
    }

    // read matrix to local memory
    for (mpFlow::dtype::index j = 0; j < density; j++) {
        // get column id
        id = columnId[threadIdx.x * mpFlow::numeric::sparseMatrix::block_size + j];
        hipDoubleComplex element = *(hipDoubleComplex*)&matrix[id + column * matrix_rows];
        hipDoubleComplex temp = id != mpFlow::dtype::invalid_index ? hipCmul(element,
            value[threadIdx.x * mpFlow::numeric::sparseMatrix::block_size + j]) : make_hipDoubleComplex(0.0f, 0.0f);

        res.x += temp.x;
        res.y += temp.y;
    }

    // set result
    result[row + column * result_rows].real(res.x);
    result[row + column * result_rows].imag(res.y);
}

// sparse matrix multiply kernel wrapper
template <
    class type
>
void mpFlow::numeric::sparseMatrixKernel::multiply(dim3 blocks, dim3 threads, hipStream_t stream,
    const type* values, const dtype::index* columnIds,
    const type* matrix, dtype::size result_rows, dtype::size matrix_rows,
    dtype::size columns, dtype::size density, type* result) {
    // call cuda kernel
    multiplyKernel<type><<<blocks, threads, 0, stream>>>(values, columnIds, matrix,
        result_rows, matrix_rows, columns, density, result);

    CudaCheckError();
}

// specialisations
// convert to sparse matrix kernel
template void mpFlow::numeric::sparseMatrixKernel::convert<float>(dim3, dim3,
    hipStream_t, const float*, mpFlow::dtype::size, mpFlow::dtype::size,
    float*, mpFlow::dtype::index*, mpFlow::dtype::index*);
template void mpFlow::numeric::sparseMatrixKernel::convert<double>(dim3, dim3,
    hipStream_t, const double*, mpFlow::dtype::size, mpFlow::dtype::size,
    double*, mpFlow::dtype::index*, mpFlow::dtype::index*);
template void mpFlow::numeric::sparseMatrixKernel::convert<thrust::complex<float> >(dim3, dim3,
    hipStream_t, const thrust::complex<float>*, mpFlow::dtype::size, mpFlow::dtype::size,
    thrust::complex<float>*, mpFlow::dtype::index*, mpFlow::dtype::index*);
template void mpFlow::numeric::sparseMatrixKernel::convert<thrust::complex<double> >(dim3, dim3,
    hipStream_t, const thrust::complex<double>*, mpFlow::dtype::size, mpFlow::dtype::size,
    thrust::complex<double>*, mpFlow::dtype::index*, mpFlow::dtype::index*);
template void mpFlow::numeric::sparseMatrixKernel::convert<mpFlow::dtype::index>(dim3, dim3,
    hipStream_t, const mpFlow::dtype::index*, mpFlow::dtype::size, mpFlow::dtype::size,
    mpFlow::dtype::index*, mpFlow::dtype::index*, mpFlow::dtype::index*);
template void mpFlow::numeric::sparseMatrixKernel::convert<int>(dim3, dim3,
    hipStream_t, const int*, mpFlow::dtype::size, mpFlow::dtype::size,
    int*, mpFlow::dtype::index*, mpFlow::dtype::index*);

// convertToMatrix kernel
template void mpFlow::numeric::sparseMatrixKernel::convertToMatrix<float>(dim3, dim3,
    hipStream_t, const float*, const mpFlow::dtype::index*,
    mpFlow::dtype::size, mpFlow::dtype::size, float*);
template void mpFlow::numeric::sparseMatrixKernel::convertToMatrix<double>(dim3, dim3,
    hipStream_t, const double*, const mpFlow::dtype::index*,
    mpFlow::dtype::size, mpFlow::dtype::size, double*);
template void mpFlow::numeric::sparseMatrixKernel::convertToMatrix<thrust::complex<float> >(dim3, dim3,
    hipStream_t, const thrust::complex<float>*, const mpFlow::dtype::index*,
    mpFlow::dtype::size, mpFlow::dtype::size, thrust::complex<float>*);
template void mpFlow::numeric::sparseMatrixKernel::convertToMatrix<thrust::complex<double> >(dim3, dim3,
    hipStream_t, const thrust::complex<double>*, const mpFlow::dtype::index*,
    mpFlow::dtype::size, mpFlow::dtype::size, thrust::complex<double>*);
template void mpFlow::numeric::sparseMatrixKernel::convertToMatrix<mpFlow::dtype::index>(dim3, dim3,
    hipStream_t, const mpFlow::dtype::index*, const mpFlow::dtype::index*,
    mpFlow::dtype::size, mpFlow::dtype::size, mpFlow::dtype::index* matrix);
template void mpFlow::numeric::sparseMatrixKernel::convertToMatrix<int>(dim3, dim3,
    hipStream_t, const int*, const mpFlow::dtype::index*,
    mpFlow::dtype::size, mpFlow::dtype::size, int* matrix);

// multiply kernel
template void mpFlow::numeric::sparseMatrixKernel::multiply<float>(dim3, dim3,
    hipStream_t, const float*, const mpFlow::dtype::index*,
    const float*, mpFlow::dtype::size, mpFlow::dtype::size,
    mpFlow::dtype::size, mpFlow::dtype::size, float*);
template void mpFlow::numeric::sparseMatrixKernel::multiply<double>(dim3, dim3,
    hipStream_t, const double*, const mpFlow::dtype::index*,
    const double*, mpFlow::dtype::size, mpFlow::dtype::size,
    mpFlow::dtype::size, mpFlow::dtype::size, double*);
template void mpFlow::numeric::sparseMatrixKernel::multiply<thrust::complex<float> >(dim3, dim3,
    hipStream_t, const thrust::complex<float>*, const mpFlow::dtype::index*,
    const thrust::complex<float>*, mpFlow::dtype::size, mpFlow::dtype::size,
    mpFlow::dtype::size, mpFlow::dtype::size, thrust::complex<float>*);
template void mpFlow::numeric::sparseMatrixKernel::multiply<thrust::complex<double> >(dim3, dim3,
    hipStream_t, const thrust::complex<double>*, const mpFlow::dtype::index*,
    const thrust::complex<double>*, mpFlow::dtype::size, mpFlow::dtype::size,
    mpFlow::dtype::size, mpFlow::dtype::size, thrust::complex<double>*);
template void mpFlow::numeric::sparseMatrixKernel::multiply<mpFlow::dtype::index>(dim3, dim3,
    hipStream_t, const mpFlow::dtype::index*, const mpFlow::dtype::index*,
    const mpFlow::dtype::index*, mpFlow::dtype::size, mpFlow::dtype::size,
    mpFlow::dtype::size, mpFlow::dtype::size, mpFlow::dtype::index*);
template void mpFlow::numeric::sparseMatrixKernel::multiply<int>(dim3, dim3,
    hipStream_t, const int*, const mpFlow::dtype::index*,
    const int*, mpFlow::dtype::size, mpFlow::dtype::size,
    mpFlow::dtype::size, mpFlow::dtype::size, int*);
