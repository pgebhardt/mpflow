#include "hip/hip_runtime.h"
// --------------------------------------------------------------------
// This file is part of mpFlow.
//
// mpFlow is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 2 of the License, or
// (at your option) any later version.
//
// mpFlow is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with mpFlow. If not, see <http://www.gnu.org/licenses/>.
//
// Copyright (C) 2014 Patrik Gebhardt
// Contact: patrik.gebhardt@rub.de
// --------------------------------------------------------------------

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <thrust/complex.h>
#include "mpflow/cuda_error.h"

#include "mpflow/dtype.h"
#include "mpflow/numeric/constants.h"
#include "mpflow/numeric/conjugate_gradient_kernel.h"

// add scalar kernel
template <
    class dataType
>
static __global__ void addScalarKernel(const dataType* scalar,
    mpFlow::dtype::size vectorRows, mpFlow::dtype::size rows,
    mpFlow::dtype::size columns, dataType* vector) {
    // get ids
    mpFlow::dtype::index row = blockIdx.x * blockDim.x + threadIdx.x;
    mpFlow::dtype::index column = blockIdx.y * blockDim.y + threadIdx.y;

    // add data
    vector[row + column * vectorRows] += row < rows && column < columns ?
        scalar[column * vectorRows] : 0.0f;
}

// add scalar kernel wrapper
template <
    class dataType
>
void mpFlow::numeric::conjugateGradientKernel::addScalar(dim3 blocks, dim3 threads,
    hipStream_t stream, const dataType* scalar, dtype::size vector_rows,
    dtype::size rows, dtype::size columns, dataType* vector) {
    // call cuda kernel
    addScalarKernel<<<blocks, threads, 0, stream>>>(scalar, vector_rows,
        rows, columns, vector);

    CudaCheckError();
}

template void mpFlow::numeric::conjugateGradientKernel::addScalar<mpFlow::dtype::real>(
    dim3, dim3, hipStream_t, const mpFlow::dtype::real*, mpFlow::dtype::size,
    mpFlow::dtype::size, mpFlow::dtype::size, mpFlow::dtype::real*);
template void mpFlow::numeric::conjugateGradientKernel::addScalar<mpFlow::dtype::complex>(
    dim3, dim3, hipStream_t, const mpFlow::dtype::complex*, mpFlow::dtype::size,
    mpFlow::dtype::size, mpFlow::dtype::size, mpFlow::dtype::complex*);

// update vector kernel
template <
    class dataType
>
static __global__ void updateVectorKernel(const dataType* x1,
    const mpFlow::dtype::real sign, const dataType* x2,
    const dataType* r1, const dataType* r2,
    mpFlow::dtype::size rows, dataType* result) {
    // get ids
    mpFlow::dtype::index row = blockIdx.x * blockDim.x + threadIdx.x;
    mpFlow::dtype::index column = blockIdx.y * blockDim.y + threadIdx.y;

    // calc value
    result[row + column * rows] = r2[column * rows] != 0.0f ? x1[row + column * rows] +
        sign * x2[row + column * rows] *
        r1[column * rows] / r2[column * rows] : 0.0f;
}

// update vector kernel wrapper
template <
    class dataType
>
void mpFlow::numeric::conjugateGradientKernel::updateVector(dim3 blocks, dim3 threads,
    hipStream_t stream, const dataType* x1, const dtype::real sign,
    const dataType* x2, const dataType* r1, const dataType* r2,
    dtype::size rows, dataType* result) {
    // call cuda kernel
    updateVectorKernel<<<blocks, threads, 0, stream>>>(x1, sign, x2, r1, r2, rows,
        result);

    CudaCheckError();
}

template void mpFlow::numeric::conjugateGradientKernel::updateVector<mpFlow::dtype::real>(
    dim3, dim3, hipStream_t, const mpFlow::dtype::real*, const mpFlow::dtype::real,
    const mpFlow::dtype::real*, const mpFlow::dtype::real*, const mpFlow::dtype::real*,
    mpFlow::dtype::size, mpFlow::dtype::real*);
template void mpFlow::numeric::conjugateGradientKernel::updateVector<mpFlow::dtype::complex>(
    dim3, dim3, hipStream_t, const mpFlow::dtype::complex*, const mpFlow::dtype::real,
    const mpFlow::dtype::complex*, const mpFlow::dtype::complex*, const mpFlow::dtype::complex*,
    mpFlow::dtype::size, mpFlow::dtype::complex*);
